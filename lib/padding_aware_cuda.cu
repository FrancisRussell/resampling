#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include "interpolate.h"
#include "padding_aware_cuda.h"
#include "timer.h"
#include "allocation.h"
#include "common.h"
#include "forward.h"
#include "fftw_cycle.h"
#include "common_cuda.h"
#include <complex.h>
#include <stdint.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

typedef enum
{
  PACKED,
  SEPARATE
} strategy_t;

/// Implementation-specific structure for padding-aware interpolation plans.
typedef struct
{
  struct interpolate_plan_s common;
  strategy_t strategy;

  hipfftHandle interleaved_forward;
  hipfftHandle n2_backward_interleaved[2];
  hipfftHandle n1_backward_interleaved[2];
  hipfftHandle n0_backward_interleaved;

  int n2_backward_interleaved_needed[2];
  int n1_backward_interleaved_needed[2];

  hipfftHandle real_forward;
  hipfftHandle n2_backward_real;
  hipfftHandle n1_backward_real;
  hipfftHandle n0_backward_real;

  int has_real_plans;
} pa_plan_s;

typedef pa_plan_s *pa_plan;

static interpolate_plan allocate_plan(void);

/* Interface functions */
static const char *get_name(const interpolate_plan plan);
static void pa_interpolate_execute_interleaved(interpolate_plan plan, rs_complex *in, rs_complex *out);
static void pa_interpolate_execute_split(interpolate_plan plan, double *rin, double *iin, double *rout, double *iout);
static void pa_interpolate_execute_split_product(interpolate_plan plan, double *rin, double *iin, double *out);
static void pa_interpolate_print_timings(const interpolate_plan plan);
static void pa_interpolate_destroy_detail(interpolate_plan plan);
static void pa_set_flags(interpolate_plan plan, const int flags);
static void pa_get_statistic_float(const interpolate_plan plan, const int statistic, const int index, stat_type_t *type, double *result);

static void plan_common(pa_plan plan, interpolation_t type, int n0, int n1, int n2, int flags);

static const char *get_name(const interpolate_plan plan)
{
  return "padding-aware-cuda";
}

static interpolate_plan allocate_plan(void)
{
  setup_threading();

  interpolate_plan holder = (interpolate_plan) malloc(sizeof(pa_plan_s));
  assert(holder != NULL);

  holder->ref_cnt = 1;

  holder->get_name = get_name;
  holder->set_flags = pa_set_flags;
  holder->get_statistic_float = pa_get_statistic_float;
  holder->execute_interleaved = pa_interpolate_execute_interleaved;
  holder->execute_split = pa_interpolate_execute_split;
  holder->execute_split_product = pa_interpolate_execute_split_product;
  holder->print_timings = pa_interpolate_print_timings;
  holder->destroy_detail = pa_interpolate_destroy_detail;

  return holder;
}

static void pa_set_flags(interpolate_plan parent, const int flags)
{
  pa_plan plan = (pa_plan) parent;

  const int conflicting_layouts = PREFER_PACKED_LAYOUT | PREFER_SPLIT_LAYOUT;
  assert((flags & conflicting_layouts) != conflicting_layouts);

  if (flags & PREFER_PACKED_LAYOUT)
    plan->strategy = PACKED;

  if (flags & PREFER_SPLIT_LAYOUT)
    plan->strategy = SEPARATE;
}

static void pa_get_statistic_float(const interpolate_plan parent, const int statistic, const int index, stat_type_t *type, double *result)
{
  *type = STATISTIC_UNKNOWN;
}

static void plan_common(pa_plan plan, interpolation_t type, int n0, int n1, int n2, int flags)
{
  populate_properties((interpolate_plan) plan, type, n0, n1, n2);
  interpolate_plan parent = cast_to_parent(plan);
  const size_t block_size = num_elements(parent);

  block_info_t fine_info;
  get_block_info_fine(parent, &fine_info);

  int rev_dims[] = { plan_input_size(parent, 2), plan_input_size(parent, 1), plan_input_size(parent, 0) };

  CUFFT_CHECK(hipfftPlanMany(&plan->interleaved_forward, 3, rev_dims, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2Z, 1));

  // For small FFT sizes, some corners may be size zero, which CUFFT dislikes.
  for(int corner = 0; corner < 2; ++corner)
  {
    plan->n2_backward_interleaved_needed[corner] = (corner_size(plan_input_size(parent, 0), corner) != 0);
    plan->n1_backward_interleaved_needed[corner] = (corner_size(plan_input_size(parent, 0), corner) != 0);
  }

  // Interpolation in direction 2, iteration in direction 0, positive frequencies
  if (plan->n2_backward_interleaved_needed[0])
  {
    CUFFT_CHECK(hipfftPlanMany(&plan->n2_backward_interleaved[0], 1, &fine_info.dims[2],
        &fine_info.strides[2], fine_info.strides[2], fine_info.strides[0],
        &fine_info.strides[2], fine_info.strides[2], fine_info.strides[0],
        HIPFFT_Z2Z, corner_size(plan_input_size(parent, 0), 0)));
  }

  // Interpolation in direction 2, iteration in direction 0, negative frequencies
  if (plan->n2_backward_interleaved_needed[1])
  {
    CUFFT_CHECK(hipfftPlanMany(&plan->n2_backward_interleaved[1], 1, &fine_info.dims[2],
        &fine_info.dims[2], fine_info.strides[2], fine_info.strides[0],
        &fine_info.dims[2], fine_info.strides[2], fine_info.strides[0],
        HIPFFT_Z2Z, corner_size(plan_input_size(parent, 0), 1)));
  }

  // Interpolation in direction 1, iteration in direction 0, positive frequencies
  if (plan->n1_backward_interleaved_needed[0])
  {
    CUFFT_CHECK(hipfftPlanMany(&plan->n1_backward_interleaved[0], 1, &fine_info.dims[1],
        &fine_info.dims[1], fine_info.strides[1], fine_info.strides[0],
        &fine_info.dims[1], fine_info.strides[1], fine_info.strides[0],
        HIPFFT_Z2Z, corner_size(plan_input_size(parent, 0), 0)));
  }

  // Interpolation in direction 1, iteration in direction 0, negative frequencies
  if (plan->n1_backward_interleaved_needed[1])
  {
    CUFFT_CHECK(hipfftPlanMany(&plan->n1_backward_interleaved[1], 1, &fine_info.dims[1],
        &fine_info.dims[1], fine_info.strides[1], fine_info.strides[0],
        &fine_info.dims[1], fine_info.strides[1], fine_info.strides[0],
        HIPFFT_Z2Z, corner_size(plan_input_size(parent, 0), 1)));
  }

  // Interpolation in direction 0, iteration in direction 1, all frequencies
  CUFFT_CHECK(hipfftPlanMany(&plan->n0_backward_interleaved, 1, &fine_info.dims[0],
      &fine_info.dims[0], fine_info.strides[0], fine_info.strides[1],
      &fine_info.dims[0], fine_info.strides[0], fine_info.strides[1],
      HIPFFT_Z2Z, fine_info.dims[1] * fine_info.dims[2]));

  plan->has_real_plans = 0;
}

interpolate_plan interpolate_plan_3d_padding_aware_cuda_interleaved(int n0, int n1, int n2, int flags)
{
  if (!has_acceptable_cuda_support())
    return NULL;

  interpolate_plan wrapper = allocate_plan();
  pa_plan plan = (pa_plan) wrapper;

  plan_common(plan, INTERPOLATE_INTERLEAVED, n0, n1, n2, flags);
  plan->strategy = PACKED;

  return wrapper;
}

interpolate_plan interpolate_plan_3d_padding_aware_cuda_split(int n0, int n1, int n2, int flags)
{
  if (!has_acceptable_cuda_support())
    return NULL;

  interpolate_plan parent = allocate_plan();
  pa_plan plan = (pa_plan) parent;

  plan_common(plan, INTERPOLATE_SPLIT, n0, n1, n2, flags);

  block_info_t coarse_info, fine_info, transformed_coarse_info, transformed_fine_info;
  get_block_info_coarse(parent, &coarse_info);
  get_block_info_fine(parent, &fine_info);
  get_block_info_real_recip_coarse(parent, &transformed_coarse_info);
  get_block_info_real_recip_fine(parent, &transformed_fine_info);

  int rev_dims[] = { plan_input_size(parent, 2), plan_input_size(parent, 1), plan_input_size(parent, 0) };

  CUFFT_CHECK(hipfftPlanMany(&plan->real_forward, 3, rev_dims, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z, 1));

  // Interpolation in direction 2, iteration in direction 0, positive frequencies
  if (plan->n2_backward_interleaved_needed[0])
  {
    CUFFT_CHECK(hipfftPlanMany(&plan->n2_backward_real, 1, &transformed_fine_info.dims[2],
      &transformed_fine_info.dims[2], transformed_fine_info.strides[2], transformed_fine_info.strides[0],
      &transformed_fine_info.dims[2], transformed_fine_info.strides[2], transformed_fine_info.strides[0],
      HIPFFT_Z2Z, corner_size(plan_input_size(parent, 0), 0)));
  }

  // Interpolation in direction 1, iteration in direction 0, positive frequencies
  if (plan->n2_backward_interleaved_needed[0])
  {
    CUFFT_CHECK(hipfftPlanMany(&plan->n1_backward_real, 1, &transformed_fine_info.dims[1],
        &transformed_fine_info.dims[1], transformed_fine_info.strides[1], transformed_fine_info.strides[0],
        &transformed_fine_info.dims[1], transformed_fine_info.strides[1], transformed_fine_info.strides[0],
        HIPFFT_Z2Z, corner_size(plan_input_size(parent, 0), 0)));
  }

  // Interpolation in direction 0, iteration in direction 1, all frequencies
  CUFFT_CHECK(hipfftPlanMany(&plan->n0_backward_real, 1, &fine_info.dims[0],
      &fine_info.dims[0], transformed_fine_info.strides[0], transformed_fine_info.strides[1],
      &fine_info.dims[0], fine_info.strides[0],             fine_info.strides[1],
      HIPFFT_Z2D, fine_info.dims[1] * fine_info.dims[2]));

  plan->has_real_plans = 1;

  plan->strategy = SEPARATE;
  const double separate_time = time_interpolate_split(parent);
  plan->strategy = PACKED;
  const double packed_time = time_interpolate_split(parent);
  plan->strategy = (separate_time < packed_time) ? SEPARATE : PACKED;

  return parent;
}

interpolate_plan interpolate_plan_3d_padding_aware_cuda_product(int n0, int n1, int n2, int flags)
{
  if (!has_acceptable_cuda_support())
    return NULL;

  interpolate_plan parent = interpolate_plan_3d_padding_aware_cuda_split(n0, n1, n2, flags);
  pa_plan plan = (pa_plan) parent;
  parent->type = INTERPOLATE_SPLIT_PRODUCT;

  plan->strategy = SEPARATE;
  const double separate_time = time_interpolate_split_product(parent);
  plan->strategy = PACKED;
  const double packed_time = time_interpolate_split_product(parent);
  plan->strategy = (separate_time < packed_time) ? SEPARATE : PACKED;

  return parent;
}

static void pa_interpolate_destroy_detail(interpolate_plan parent)
{
  pa_plan plan = (pa_plan) parent;

  hipfftDestroy(plan->interleaved_forward);

  for(int corner = 0; corner < 2; ++corner)
  {
    if (plan->n2_backward_interleaved_needed[corner])
      hipfftDestroy(plan->n2_backward_interleaved[corner]);

    if (plan->n1_backward_interleaved_needed[corner])
      hipfftDestroy(plan->n1_backward_interleaved[corner]);
  }

  hipfftDestroy(plan->n0_backward_interleaved);

  if (plan->has_real_plans)
  {
    hipfftDestroy(plan->real_forward);
    hipfftDestroy(plan->n2_backward_real);
    hipfftDestroy(plan->n1_backward_real);
    hipfftDestroy(plan->n0_backward_real);
  }
}

static void backward_transform_c2c(const pa_plan plan, const block_info_t *data_info, hipDoubleComplex *data)
{
  size_t corner_sizes[3][2];
  interpolate_plan parent = cast_to_parent(plan);

  for(int negative = 0; negative < 2; ++negative)
    for(int dim = 0; dim < 3; ++dim)
      corner_sizes[dim][negative] = corner_size(plan_input_size(parent, dim), negative);

  // Interpolation in direction 2
  for(size_t y = 0; y < corner_sizes[1][0]; ++y)
  {
    hipDoubleComplex *positive_start = data + y * data_info->strides[1];
    if (plan->n2_backward_interleaved_needed[0])
      CUFFT_CHECK(hipfftExecZ2Z(plan->n2_backward_interleaved[0], positive_start, positive_start, HIPFFT_BACKWARD));

    hipDoubleComplex *negative_start = data + (y + 1) * data_info->strides[1] - corner_sizes[0][1];
    if (plan->n2_backward_interleaved_needed[1])
      CUFFT_CHECK(hipfftExecZ2Z(plan->n2_backward_interleaved[1], negative_start, negative_start, HIPFFT_BACKWARD));
  }

  for(size_t y = data_info->dims[1] - corner_sizes[1][1]; y < data_info->dims[1]; ++y)
  {
    hipDoubleComplex *positive_start = data + y * data_info->strides[1];
    if (plan->n2_backward_interleaved_needed[0])
      CUFFT_CHECK(hipfftExecZ2Z(plan->n2_backward_interleaved[0], positive_start, positive_start, HIPFFT_BACKWARD));

    hipDoubleComplex *negative_start = data + (y + 1) * data_info->strides[1] - corner_sizes[0][1];
    if (plan->n2_backward_interleaved_needed[1])
      CUFFT_CHECK(hipfftExecZ2Z(plan->n2_backward_interleaved[1], negative_start, negative_start, HIPFFT_BACKWARD));
  }

  // Interpolation in direction 1
  for(size_t z = 0; z < data_info->dims[2]; ++z)
  {
    hipDoubleComplex *positive_start = data + z * data_info->strides[2];
    if (plan->n1_backward_interleaved_needed[0])
      CUFFT_CHECK(hipfftExecZ2Z(plan->n1_backward_interleaved[0], positive_start, positive_start, HIPFFT_BACKWARD));

    hipDoubleComplex *negative_start = data + z * data_info->strides[2] + data_info->strides[1] - corner_sizes[0][1];
    if (plan->n1_backward_interleaved_needed[1])
      CUFFT_CHECK(hipfftExecZ2Z(plan->n1_backward_interleaved[1], negative_start, negative_start, HIPFFT_BACKWARD));
  }

  // Interpolation in direction 0
  CUFFT_CHECK(hipfftExecZ2Z(plan->n0_backward_interleaved, data, data, HIPFFT_BACKWARD));
}

static void backward_transform_c2r(const pa_plan plan,
  const block_info_t *from_info, hipDoubleComplex *from,
  const block_info_t *to_info, double *to)
{
  size_t corner_sizes[3][2];
  interpolate_plan parent = cast_to_parent(plan);

  for(int negative = 0; negative < 2; ++negative)
    for(int dim = 0; dim < 3; ++dim)
      corner_sizes[dim][negative] = corner_size(plan_input_size(parent, dim), negative);

  // Interpolation in direction 2
  for(size_t y = 0; y < corner_sizes[1][0]; ++y)
  {
    hipDoubleComplex *positive_start = from + y * from_info->strides[1];
    if (plan->n2_backward_interleaved_needed[0])
      CUFFT_CHECK(hipfftExecZ2Z(plan->n2_backward_real, positive_start, positive_start, HIPFFT_BACKWARD));
  }

  for(size_t y = to_info->dims[1] - corner_sizes[1][1]; y < to_info->dims[1]; ++y)
  {
    hipDoubleComplex *positive_start = from + y * from_info->strides[1];
    if (plan->n2_backward_interleaved_needed[0])
      CUFFT_CHECK(hipfftExecZ2Z(plan->n2_backward_real, positive_start, positive_start, HIPFFT_BACKWARD));
  }

  // Interpolation in direction 1
  for(size_t z = 0; z < from_info->dims[2]; ++z)
  {
    hipDoubleComplex *positive_start = from + z * from_info->strides[2];
    if (plan->n1_backward_interleaved_needed[0])
      CUFFT_CHECK(hipfftExecZ2Z(plan->n1_backward_real, positive_start, positive_start, HIPFFT_BACKWARD));
  }

  // Interpolation in direction 0
  CUFFT_CHECK(hipfftExecZ2D(plan->n0_backward_real, from, to));
}

static void pa_interpolate_execute_interleaved(interpolate_plan parent, rs_complex *in, rs_complex *out)
{
  pa_plan plan = (pa_plan) parent;
  assert(plan->strategy == PACKED);

  block_info_t coarse_info, fine_info;
  get_block_info_coarse(parent, &coarse_info);
  get_block_info_fine(parent, &fine_info);
  const size_t block_size = num_elements_block(&coarse_info);

  thrust::device_vector<hipDoubleComplex> dev_in(block_size);
  thrust::device_vector<hipDoubleComplex> dev_out(block_size * 8);

  CUDA_CHECK(hipHostRegister(in, sizeof(rs_complex) * block_size, 0));
  CUDA_CHECK(hipMemcpy(thrust::raw_pointer_cast(&dev_in[0]), in, sizeof(rs_complex) * block_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipHostUnregister(in));

  CUFFT_CHECK(hipfftExecZ2Z(plan->interleaved_forward, thrust::raw_pointer_cast(&dev_in[0]), thrust::raw_pointer_cast(&dev_in[0]), HIPFFT_FORWARD));

  halve_nyquist_components_cuda(parent, &coarse_info, thrust::raw_pointer_cast(&dev_in[0]));
  pad_coarse_to_fine_interleaved_cuda(parent,
    &coarse_info, thrust::raw_pointer_cast(&dev_in[0]), &fine_info, thrust::raw_pointer_cast(&dev_out[0]), 0);

  backward_transform_c2c(plan, &fine_info, thrust::raw_pointer_cast(&dev_out[0]));

  CUDA_CHECK(hipHostRegister(out, sizeof(rs_complex) * block_size * 8, 0));
  CUDA_CHECK(hipMemcpy(out, thrust::raw_pointer_cast(&dev_out[0]), sizeof(rs_complex) * block_size * 8, hipMemcpyDeviceToHost));
  CUDA_CHECK(hipHostUnregister(out));
  CUDA_CHECK(hipDeviceSynchronize());
}

static void pa_interpolate_real(pa_plan plan, double *in, const thrust::device_ptr<double>& dev_out)
{
  block_info_t coarse_info, fine_info, transformed_coarse_info, transformed_fine_info;
  interpolate_plan parent = cast_to_parent(plan);
  get_block_info_coarse(parent, &coarse_info);
  get_block_info_fine(parent, &fine_info);
  get_block_info_real_recip_coarse(parent, &transformed_coarse_info);
  get_block_info_real_recip_fine(parent, &transformed_fine_info);

  const size_t block_size = num_elements_block(&coarse_info);
  const size_t transformed_size_coarse = num_elements_block(&transformed_coarse_info);
  const size_t transformed_size_fine = num_elements_block(&transformed_fine_info);

  thrust::device_vector<double> dev_in(block_size);
  thrust::device_vector<hipDoubleComplex> scratch_coarse(transformed_size_coarse);
  thrust::device_vector<hipDoubleComplex> scratch_fine(transformed_size_fine);

  CUDA_CHECK(hipHostRegister(in, sizeof(double) * block_size, 0));
  CUDA_CHECK(hipMemcpy(thrust::raw_pointer_cast(&dev_in[0]), in, sizeof(double) * block_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipHostUnregister(in));

  CUFFT_CHECK(hipfftExecD2Z(plan->real_forward, thrust::raw_pointer_cast(&dev_in[0]), thrust::raw_pointer_cast(&scratch_coarse[0])));

  halve_nyquist_components_cuda(parent, &transformed_coarse_info, thrust::raw_pointer_cast(&scratch_coarse[0]));
  pad_coarse_to_fine_interleaved_cuda(parent,
    &transformed_coarse_info, thrust::raw_pointer_cast(&scratch_coarse[0]),
    &transformed_fine_info,   thrust::raw_pointer_cast(&scratch_fine[0]), 1);

  backward_transform_c2r(plan, &transformed_fine_info, thrust::raw_pointer_cast(&scratch_fine[0]),
    &fine_info, thrust::raw_pointer_cast(dev_out));
}

static void pa_interpolate_execute_split(interpolate_plan parent, double *rin, double *iin, double *rout, double *iout)
{
  pa_plan plan = (pa_plan) parent;
  assert(INTERPOLATE_SPLIT == parent->type || INTERPOLATE_SPLIT_PRODUCT == parent->type);

  if (plan->strategy == PACKED)
  {
    block_info_t coarse_info, fine_info;
    get_block_info_coarse(parent, &coarse_info);
    get_block_info_fine(parent, &fine_info);
    const size_t block_size = num_elements_block(&coarse_info);

    rs_complex *const scratch_coarse = rs_alloc_complex(block_size);
    rs_complex *const scratch_fine = rs_alloc_complex(8 * block_size);

    interleave_real(block_size, (double*) scratch_coarse, rin, iin);
    pa_interpolate_execute_interleaved(parent, scratch_coarse, scratch_fine);
    deinterleave_real(8 * block_size, (const double*) scratch_fine, rout, iout);

    rs_free(scratch_fine);
    rs_free(scratch_coarse);
  }
  else if (plan->strategy == SEPARATE)
  {
    block_info_t fine_info;
    get_block_info_fine(parent, &fine_info);
    const size_t fine_block_size = num_elements_block(&fine_info);

    thrust::device_vector<double> dev_out_r(fine_block_size);
    thrust::device_vector<double> dev_out_i(fine_block_size);

    pa_interpolate_real(plan, rin, &dev_out_r[0]);
    pa_interpolate_real(plan, iin, &dev_out_i[0]);

    CUDA_CHECK(hipHostRegister(rout, sizeof(double) * fine_block_size, 0));
    CUDA_CHECK(hipMemcpy(rout, thrust::raw_pointer_cast(&dev_out_r[0]), sizeof(double) * fine_block_size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipHostUnregister(rout));

    CUDA_CHECK(hipHostRegister(iout, sizeof(double) * fine_block_size, 0));
    CUDA_CHECK(hipMemcpy(iout, thrust::raw_pointer_cast(&dev_out_i[0]), sizeof(double) * fine_block_size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipHostUnregister(iout));

    CUDA_CHECK(hipDeviceSynchronize());
  }
  else
  {
    assert(0 && "Unkown strategy");
  }
}

static void pa_interpolate_execute_split_product(interpolate_plan parent, double *rin, double *iin, double *out)
{
  pa_plan plan = (pa_plan) parent;
  assert(INTERPOLATE_SPLIT_PRODUCT == parent->type);

  const size_t block_size = num_elements(parent);

  if (plan->strategy == PACKED)
  {
    rs_complex *const scratch_coarse = rs_alloc_complex(block_size);
    rs_complex *const scratch_fine = rs_alloc_complex(8 * block_size);

    interleave_real(block_size, (double*) scratch_coarse, rin, iin);
    pa_interpolate_execute_interleaved(parent, scratch_coarse, scratch_fine);
    complex_to_product(8 * block_size, scratch_fine, out);

    rs_free(scratch_coarse);
    rs_free(scratch_fine);
  }
  else if (plan->strategy == SEPARATE)
  {
    block_info_t fine_info;
    get_block_info_fine(parent, &fine_info);
    const size_t fine_block_size = num_elements_block(&fine_info);

    thrust::device_vector<double> dev_out_r(fine_block_size);
    thrust::device_vector<double> dev_out_i(fine_block_size);

    pa_interpolate_real(plan, rin, &dev_out_r[0]);
    pa_interpolate_real(plan, iin, &dev_out_i[0]);

    thrust::transform(dev_out_r.begin(), dev_out_r.end(), dev_out_i.begin(), dev_out_r.begin(), thrust::plus<double>());

    CUDA_CHECK(hipHostRegister(out, sizeof(double) * block_size * 8, 0));
    CUDA_CHECK(hipMemcpy(out, thrust::raw_pointer_cast(&dev_out_r[0]), sizeof(double) * block_size * 8, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipHostUnregister(out));
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

static void pa_interpolate_print_timings(const interpolate_plan plan)
{
}
