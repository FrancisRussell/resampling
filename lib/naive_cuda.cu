#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include "tintl/naive_cuda.h"
#include "tintl/interpolate.h"
#include "tintl/allocation.h"
#include "tintl/timer.h"
#include "common.h"
#include "tintl/forward.h"
#include "common_cuda.h"
#include <complex.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <assert.h>

typedef enum
{
  PACKED,
  SEPARATE
} strategy_t;


/// Implementation-specific structure for naive interpolation plans.
typedef struct
{
  struct interpolate_plan_s common;
  strategy_t strategy;

  hipfftHandle interleaved_forward;
  hipfftHandle interleaved_backward;

  int has_real_plans;
  hipfftHandle real_forward;
  hipfftHandle real_backward;
} naive_plan_s;


typedef naive_plan_s *naive_plan;

static interpolate_plan allocate_plan(void);

/* Interface functions */


static const char *get_name(const interpolate_plan plan);
static void naive_interpolate_execute_interleaved(interpolate_plan plan, rs_complex *in, rs_complex *out);
static void naive_interpolate_execute_split(interpolate_plan plan, double *rin, double *iin, double *rout, double *iout);
static void naive_interpolate_execute_split_product(interpolate_plan plan, double *rin, double *iin, double *out);
static void naive_interpolate_print_timings(const interpolate_plan plan);
static void naive_interpolate_destroy_detail(interpolate_plan plan);
static void naive_set_flags(interpolate_plan plan, const int flags);
static void naive_get_statistic_float(const interpolate_plan plan, int statistic, int index, stat_type_t *type, double *result);

static void plan_common(naive_plan plan, interpolation_t type, int n0, int n1, int n2, int flags);

static const char *get_name(const interpolate_plan plan)
{
  return "naive-cuda";
}

static interpolate_plan allocate_plan(void)
{
  setup_threading();

  interpolate_plan holder = (interpolate_plan) malloc(sizeof(naive_plan_s));
  assert(holder != NULL);

  holder->get_name = get_name;
  holder->set_flags = naive_set_flags;
  holder->get_statistic_float = naive_get_statistic_float;
  holder->execute_interleaved = naive_interpolate_execute_interleaved;
  holder->execute_split = naive_interpolate_execute_split;
  holder->execute_split_product = naive_interpolate_execute_split_product;
  holder->print_timings = naive_interpolate_print_timings;
  holder->destroy_detail = naive_interpolate_destroy_detail;

  return holder;
}

static void naive_set_flags(interpolate_plan parent, const int flags)
{
  naive_plan plan = (naive_plan) parent;

  const int conflicting_layouts = PREFER_PACKED_LAYOUT | PREFER_SPLIT_LAYOUT;
  assert((flags & conflicting_layouts) != conflicting_layouts);

  if (flags & PREFER_PACKED_LAYOUT)
    plan->strategy = PACKED;

  if (flags & PREFER_SPLIT_LAYOUT)
    plan->strategy = SEPARATE;
}

static void naive_get_statistic_float(const interpolate_plan parent, int statistic, int index, stat_type_t *type, double *result)
{
  *type = STATISTIC_UNKNOWN;
}

static void plan_common(naive_plan plan, interpolation_t type, int n0, int n1, int n2, int flags)
{
  populate_properties((interpolate_plan) plan, type, n0, n1, n2);
  interpolate_plan parent = cast_to_parent(plan);

  block_info_t coarse_info, fine_info;
  get_block_info_coarse(parent, &coarse_info);
  get_block_info_fine(parent, &fine_info);

  int rev_dims[] = { coarse_info.dims[2], coarse_info.dims[1], coarse_info.dims[0] };
  int rev_fine_dims[] = { fine_info.dims[2], fine_info.dims[1], fine_info.dims[0] };

  CUFFT_CHECK(hipfftPlanMany(&plan->interleaved_forward, 3, rev_dims, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2Z, 1));
  CUFFT_CHECK(hipfftPlanMany(&plan->interleaved_backward, 3, rev_fine_dims, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2Z, 1));

  plan->has_real_plans = 0;
}

interpolate_plan interpolate_plan_3d_naive_cuda_interleaved(int n0, int n1, int n2, int flags)
{
  if (!has_acceptable_cuda_support())
    return NULL;

  interpolate_plan wrapper = allocate_plan();
  naive_plan plan = (naive_plan) wrapper;

  plan_common(plan, INTERPOLATE_INTERLEAVED, n0, n1, n2, flags);
  plan->strategy = PACKED;

  return wrapper;
}

interpolate_plan interpolate_plan_3d_naive_cuda_split(int n0, int n1, int n2, int flags)
{
  if (!has_acceptable_cuda_support())
    return NULL;

  interpolate_plan parent = allocate_plan();
  naive_plan plan = (naive_plan) parent;

  plan_common(plan, INTERPOLATE_SPLIT, n0, n1, n2, flags);

  block_info_t coarse_info, fine_info, transformed_coarse_info, transformed_fine_info;
  get_block_info_coarse(parent, &coarse_info);
  get_block_info_fine(parent, &fine_info);
  get_block_info_real_recip_coarse(parent, &transformed_coarse_info);
  get_block_info_real_recip_fine(parent, &transformed_fine_info);

  int rev_dims[] = { coarse_info.dims[2], coarse_info.dims[1], coarse_info.dims[0] };
  int rev_fine_dims[] = { fine_info.dims[2], fine_info.dims[1], fine_info.dims[0] };

  CUFFT_CHECK(hipfftPlanMany(&plan->real_forward, 3, rev_dims, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z, 1));
  CUFFT_CHECK(hipfftPlanMany(&plan->real_backward, 3, rev_fine_dims, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D, 1));

  plan->strategy = SEPARATE;
  const double separate_time = time_interpolate_split(parent);
  plan->strategy = PACKED;
  const double packed_time = time_interpolate_split(parent);
  plan->strategy = (separate_time < packed_time) ? SEPARATE : PACKED;

  plan->has_real_plans = 1;

  return parent;
}

interpolate_plan interpolate_plan_3d_naive_cuda_product(int n0, int n1, int n2, int flags)
{
  if (!has_acceptable_cuda_support())
    return NULL;

  interpolate_plan parent = interpolate_plan_3d_naive_cuda_split(n0, n1, n2, flags);
  parent->type = INTERPOLATE_SPLIT_PRODUCT;
  naive_plan plan = (naive_plan) parent;

  plan->strategy = SEPARATE;
  const double separate_time = time_interpolate_split_product(parent);
  plan->strategy = PACKED;
  const double packed_time = time_interpolate_split_product(parent);
  plan->strategy = (separate_time < packed_time) ? SEPARATE : PACKED;
  return parent;
}

static void naive_interpolate_destroy_detail(interpolate_plan parent)
{
  naive_plan plan = (naive_plan) parent;

  hipfftDestroy(plan->interleaved_forward);
  hipfftDestroy(plan->interleaved_backward);

  if (plan->has_real_plans)
  {
    hipfftDestroy(plan->real_forward);
    hipfftDestroy(plan->real_backward);
  }
}

static void naive_interpolate_execute_interleaved(interpolate_plan parent, rs_complex *in, rs_complex *out)
{
  naive_plan plan = (naive_plan) parent;
  assert(plan->strategy == PACKED);

  block_info_t coarse_info, fine_info;
  get_block_info_coarse(parent, &coarse_info);
  get_block_info_fine(parent, &fine_info);
  const size_t block_size = num_elements_block(&coarse_info);

  thrust::device_vector<hipDoubleComplex> dev_in(block_size);
  thrust::device_vector<hipDoubleComplex> dev_out(block_size * 8);

  CUDA_CHECK(hipHostRegister(in, sizeof(rs_complex) * block_size, 0));
  CUDA_CHECK(hipMemcpy(thrust::raw_pointer_cast(&dev_in[0]), in, sizeof(rs_complex) * block_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipHostUnregister(in));

  CUFFT_CHECK(hipfftExecZ2Z(plan->interleaved_forward, thrust::raw_pointer_cast(&dev_in[0]), thrust::raw_pointer_cast(&dev_in[0]), HIPFFT_FORWARD));

  halve_nyquist_components_cuda(parent, &coarse_info, thrust::raw_pointer_cast(&dev_in[0]));
  pad_coarse_to_fine_interleaved_cuda(parent,
    &coarse_info, thrust::raw_pointer_cast(&dev_in[0]), &fine_info, thrust::raw_pointer_cast(&dev_out[0]), 0);

  CUFFT_CHECK(hipfftExecZ2Z(plan->interleaved_backward,
    thrust::raw_pointer_cast(&dev_out[0]), thrust::raw_pointer_cast(&dev_out[0]), HIPFFT_BACKWARD));

  CUDA_CHECK(hipHostRegister(out, sizeof(rs_complex) * block_size * 8, 0));
  CUDA_CHECK(hipMemcpy(out, thrust::raw_pointer_cast(&dev_out[0]), sizeof(rs_complex) * block_size * 8, hipMemcpyDeviceToHost));
  CUDA_CHECK(hipHostUnregister(out));

  CUDA_CHECK(hipDeviceSynchronize());
}

static void naive_interpolate_real(naive_plan plan, double *in, const thrust::device_ptr<double>& dev_out)
{
  block_info_t coarse_info, transformed_coarse_info, transformed_fine_info;
  interpolate_plan parent = cast_to_parent(plan);
  get_block_info_coarse(parent, &coarse_info);
  get_block_info_real_recip_coarse(parent, &transformed_coarse_info);
  get_block_info_real_recip_fine(parent, &transformed_fine_info);

  const size_t block_size = num_elements_block(&coarse_info);
  const size_t transformed_size_coarse = num_elements_block(&transformed_coarse_info);
  const size_t transformed_size_fine = num_elements_block(&transformed_fine_info);

  thrust::device_vector<double> dev_in(block_size);
  thrust::device_vector<hipDoubleComplex> scratch_coarse(transformed_size_coarse);
  thrust::device_vector<hipDoubleComplex> scratch_fine(transformed_size_fine);

  CUDA_CHECK(hipHostRegister(in, sizeof(double) * block_size, 0));
  CUDA_CHECK(hipMemcpy(thrust::raw_pointer_cast(&dev_in[0]), in, sizeof(double) * block_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipHostUnregister(in));

  CUFFT_CHECK(hipfftExecD2Z(plan->real_forward, thrust::raw_pointer_cast(&dev_in[0]), thrust::raw_pointer_cast(&scratch_coarse[0])));

  halve_nyquist_components_cuda(parent, &transformed_coarse_info, thrust::raw_pointer_cast(&scratch_coarse[0]));
  pad_coarse_to_fine_interleaved_cuda(parent,
    &transformed_coarse_info, thrust::raw_pointer_cast(&scratch_coarse[0]), &transformed_fine_info, thrust::raw_pointer_cast(&scratch_fine[0]), 1);

  CUFFT_CHECK(hipfftExecZ2D(plan->real_backward, thrust::raw_pointer_cast(&scratch_fine[0]), thrust::raw_pointer_cast(dev_out)));
}

static void naive_interpolate_execute_split(interpolate_plan parent, double *rin, double *iin, double *rout, double *iout)
{
  naive_plan plan = (naive_plan) parent;
  assert(INTERPOLATE_SPLIT == parent->type || INTERPOLATE_SPLIT_PRODUCT == parent->type);

  if (plan->strategy == PACKED)
  {
    block_info_t coarse_info, fine_info;
    get_block_info_coarse(parent, &coarse_info);
    get_block_info_fine(parent, &fine_info);
    const size_t block_size = num_elements_block(&coarse_info);

    rs_complex *const scratch_coarse = tintl_alloc_complex(block_size);
    rs_complex *const scratch_fine = tintl_alloc_complex(8 * block_size);
    assert(scratch_coarse != NULL);
    assert(scratch_fine != NULL);

    interleave_real(block_size, (double*) scratch_coarse, rin, iin);
    naive_interpolate_execute_interleaved(parent, scratch_coarse, scratch_fine);
    deinterleave_real(8 * block_size, (const double*) scratch_fine, rout, iout);

    tintl_free(scratch_fine);
    tintl_free(scratch_coarse);
  }
  else if (plan->strategy == SEPARATE)
  {
    block_info_t coarse_info;
    get_block_info_coarse(parent, &coarse_info);
    const size_t block_size = num_elements_block(&coarse_info);

    CUDA_CHECK(hipHostRegister(rout, sizeof(double) * block_size * 8, 0));
    CUDA_CHECK(hipHostRegister(iout, sizeof(double) * block_size * 8, 0));

    thrust::device_vector<double> dev_out_r(block_size * 8);
    thrust::device_vector<double> dev_out_i(block_size * 8);

    naive_interpolate_real(plan, rin, &dev_out_r[0]);
    naive_interpolate_real(plan, iin, &dev_out_i[0]);

    CUDA_CHECK(hipMemcpy(rout, thrust::raw_pointer_cast(&dev_out_r[0]), sizeof(double) * block_size * 8, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(iout, thrust::raw_pointer_cast(&dev_out_i[0]), sizeof(double) * block_size * 8, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipHostUnregister(rout));
    CUDA_CHECK(hipHostUnregister(iout));
    CUDA_CHECK(hipDeviceSynchronize());
  }
  else
  {
    assert(0 && "Unknown strategy.");
  }
}

static void naive_interpolate_execute_split_product(interpolate_plan parent, double *rin, double *iin, double *out)
{
  naive_plan plan = (naive_plan) parent;
  assert(INTERPOLATE_SPLIT_PRODUCT == parent->type);
  const size_t block_size = num_elements(parent);

  if (plan->strategy == PACKED)
  {
    rs_complex *const scratch_coarse = tintl_alloc_complex(block_size);
    rs_complex *const scratch_fine = tintl_alloc_complex(8 * block_size);
    assert(scratch_coarse != NULL);
    assert(scratch_fine != NULL);

    interleave_real(block_size, (double*) scratch_coarse, rin, iin);
    naive_interpolate_execute_interleaved(parent, scratch_coarse, scratch_fine);
    complex_to_product(8 * block_size, scratch_fine, out);

    tintl_free(scratch_coarse);
    tintl_free(scratch_fine);
  }
  if (plan->strategy == SEPARATE)
  {
    block_info_t coarse_info;
    get_block_info_coarse(parent, &coarse_info);
    const size_t block_size = num_elements_block(&coarse_info);

    CUDA_CHECK(hipHostRegister(rin, sizeof(double) * block_size, 0));
    CUDA_CHECK(hipHostRegister(iin, sizeof(double) * block_size, 0));
    CUDA_CHECK(hipHostRegister(out, sizeof(double) * block_size * 8, 0));

    thrust::device_vector<double> dev_out_r(block_size * 8);
    thrust::device_vector<double> dev_out_i(block_size * 8);

    naive_interpolate_real(plan, rin, &dev_out_r[0]);
    naive_interpolate_real(plan, iin, &dev_out_i[0]);

    CUDA_CHECK(hipHostUnregister(rin));
    CUDA_CHECK(hipHostUnregister(iin));

    thrust::transform(dev_out_r.begin(), dev_out_r.end(), dev_out_i.begin(), dev_out_r.begin(), thrust::plus<double>());

    CUDA_CHECK(hipMemcpy(out, thrust::raw_pointer_cast(&dev_out_r[0]), sizeof(double) * block_size * 8, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipHostUnregister(out));
    CUDA_CHECK(hipDeviceSynchronize());
  }
  else
  {
    assert(0 && "Unknown strategy");
  }
}

static void naive_interpolate_print_timings(const interpolate_plan parent)
{
}

